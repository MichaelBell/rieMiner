#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <mutex>

#include "prime-gmp.h"

#include "primetestcxt.h"

typedef uint32_t uint;
typedef uint64_t ulong;

#define MAX_JOB_SIZE MAX_JOB_SIZE_PRIME
#define MAX_BLOCK_SIZE 64

template<uint N_Size>
__device__
void squareSimple(uint* P, const uint* R) {
	const uint highbit = ((uint)1) << 31;

	uint T[(N_Size - 1) * 2];

	{
		uint cy = 0;
		for (int i = 0; i < N_Size - 1; ++i)
		{
			ulong p = ulong(R[i + 1]) * ulong(R[0]) + cy;
			T[i] = uint(p);
			cy = uint(p >> 32);
		}
		T[N_Size - 1] = cy;
	}

#pragma unroll 1
	for (int j = 2; j < N_Size; ++j)
	{
		uint cy = 0;
		for (int i = j; i < N_Size; ++i)
		{
			ulong p = ulong(R[i]) * ulong(R[j - 1]);
			p += cy;
			p += T[i + j - 2];
			T[i + j - 2] = uint(p);
			cy = uint(p >> 32);
		}
		T[N_Size + j - 2] = cy;
	}

	// Better not to include this into the next loop as doing it first
	// avoids latency stalls.
	for (int i = 0; i < N_Size; ++i)
	{
		P[2 * i] = R[i] * R[i];
		P[2 * i + 1] = __umulhi(R[i], R[i]);
	}

	uint cy = 0;
	for (int i = 0; i < N_Size - 1; ++i)
	{
		uint t = T[2 * i] & highbit;
		ulong a = ulong(P[2 * i + 1]) + cy;
		a += T[2 * i] << 1;
		P[2 * i + 1] = uint(a);
		cy = (t >> 31) + uint(a >> 32);

		t = T[2 * i + 1] & highbit;
		a = ulong(P[2 * i + 2]) + cy;
		a += T[2 * i + 1] << 1;
		P[2 * i + 2] = uint(a);
		cy = (t >> 31) + uint(a >> 32);
	}
	P[2 * N_Size - 1] += cy;
}

template<uint N_Size>
__device__
uint bigAdd(uint* R, const uint* A, const uint* B)
{
        uint cy = 0;
        for (int i = 0; i < N_Size; ++i)
        {
                ulong a = (ulong)A[i] + (ulong)B[i];
                a += cy;
                R[i] = (uint)a;
                cy = (uint)(a >> 32);
        }
        return cy;
}

template<uint N_Size>
__device__
int32_t bigSub(uint* R, const uint* A, const uint* B)
{
        int32_t cy = 0;
        for (int i = 0; i < N_Size; ++i)
        {
                int64_t a = (int64_t)A[i] - (int64_t)B[i];
                a += cy;
                R[i] = (uint)a;
                cy = (int32_t)(a >> 32);
        }
        return cy;
}

template<uint N_Size>
__device__
bool lessThan(const uint* A, const uint* B)
{
        for (int i = N_Size - 1; i >= 0; --i)
        {
                if (A[i] < B[i]) return true;
                if (A[i] > B[i]) return false;
        }
        return false;
}

template<uint N_Size>
__device__
void toom2SquareFull(uint* P, const uint* R)
{
        constexpr uint s = N_Size >> 1;
        constexpr uint n = N_Size - s;  // n==s or n==s+1

        const uint* a0 = R;
        const uint* a1 = R + n;
        uint* asm1 = P;

        if (s == n)
        {
                if (lessThan<n>(a0, a1))
                        bigSub<n>(asm1, a1, a0);
                else
                        bigSub<n>(asm1, a0, a1);
        }
        else
        {
                if (a0[s] == 0 && lessThan<s>(a0, a1))
                {
                        bigSub<s>(asm1, a1, a0);
                        asm1[s] = 0;
                }
                else
                {
                        asm1[s] = a0[s] + bigSub<s>(asm1, a0, a1);
                }
                P[4*n-2] = 0;
                P[4*n-1] = 0;
        }

        uint* v0 = P;
        uint vm1[N_Size + 1];
        uint* vinf = P + 2 * n;

        squareSimple<n>(vm1, asm1);
        squareSimple<s>(vinf, a1);
        squareSimple<n>(v0, a0);
        //toom2SquareHalf(vm1, asm1, n);
        //toom2SquareHalf(vinf, a1, s);
        //toom2SquareHalf(v0, a0, n);

        int32_t cy  = bigAdd<n>(P + 2*n, v0 + n, vinf);
        uint cy2 = cy + bigAdd<n>(P + n, P + 2*n, v0);
        cy += bigAdd<n>(P + 2*n, P + 2*n, vinf + n);
        cy += bigSub<2*n>(P + n, P + n, vm1);

        for (int i = 2*n; cy2 > 0; ++i)
        {
                ulong a = (ulong)P[i] + cy2;
                P[i] = (uint)a;
                cy2 = (uint)(a >> 32);
        }
        for (int i = 3*n; cy != 0; ++i)
        {
                int64_t a = (int64_t)P[i] + cy;
                P[i] = (uint)a;
                cy = (int32_t)(a >> 32);
        }
}

template<uint N_Size>
__global__
void fermat_test(const uint *M_in, const uint *Mi_in, const uint *R_in, uint *is_prime) {

	uint R[N_Size];
	uint M[N_Size];

	{
		// Get the index of the current element to be processed
		const int offset = (blockDim.x*blockIdx.x + threadIdx.x) * N_Size;

		for (int i = 0; i < N_Size; ++i)
		{
			M[i] = M_in[offset + i];
			R[i] = R_in[offset + i];
		}
	}

	const uint shift = __clz(M[N_Size - 1]);
	const uint highbit = ((uint)1) << 31;
	uint startbit;
	int en = N_Size;

        if (shift < 24)
        {
                startbit = highbit >> (shift + 8);
        }
        else
        {
                startbit = highbit >> (shift - 24);
                en--;
        }

	const uint mi = Mi_in[blockDim.x*blockIdx.x + threadIdx.x];

#pragma unroll 1
	while (en-- > 0)
	{
		uint bit = startbit;
		startbit = highbit;
		uint E = M[en];
		if (en == 0) E--;

		do
		{
			{
				uint P[N_Size * 2 + 1];
				//mpn_sqr(pp, rp, mn);
				squareSimple<N_Size>(P, R);
				//toom2SquareFull<N_Size>(P, R);

				//if (mpn_redc_1(rp, pp, mp, mn, mi) != 0) 
				//  mpn_sub_n(rp, rp, mshifted, n);
#pragma unroll 1
				for (int j = 0; j < N_Size; ++j)
				{
					uint cy = 0;
					uint v = P[j] * mi;
					for (int i = 0; i < N_Size; ++i)
					{
						ulong p = ulong(M[i]) * ulong(v) + cy;
						p += P[i + j];
						P[i + j] = uint(p);
						cy = uint(p >> 32);
					}
					R[j] = cy;
				}

				{
					uint cy = 0;
					for (int i = 0; i < N_Size; ++i)
					{
						ulong a = ulong(R[i]) + cy;
						a += P[i + N_Size];
						R[i] = uint(a);
						cy = uint(a >> 32);
					}

					if (cy != 0)
					{
						int32_t borrow = 0;
						uint last_shifted = 0;
						for (int i = 0; i < N_Size; ++i)
						{
							int64_t a = R[i];
							uint b = (M[i] << shift) | last_shifted;
							last_shifted = M[i] >> (32 - shift);
							a = a - int64_t(b) + borrow;
							R[i] = uint(a);
							borrow = int32_t(a >> 32);
						}
					}
				}
			}

			if (E & bit)
			{
				//mp_limb_t carry = mpn_lshift(rp, rp, mn, 1);
				uint carry = 0;
				for (int i = 0; i < N_Size; ++i)
				{
					uint t = R[i] & highbit;
					R[i] <<= 1;
					R[i] |= carry;
					carry = t >> 31;
				}
				while (carry)
				{
					//carry -= mpn_sub_n(rp, rp, mshifted, mn);
					int32_t borrow = 0;
					uint last_shifted = 0;
					for (int i = 0; i < N_Size; ++i)
					{
						int64_t a = R[i];
						uint b = (M[i] << shift) | last_shifted;
						last_shifted = M[i] >> (32 - shift);
						a = a - int64_t(b) + borrow;
						R[i] = uint(a);
						borrow = int32_t(a >> 32);
					}
					carry += borrow;
				}
			}
			bit >>= 1;
		} while (bit > 0);

	}

	// DeREDCify - necessary as rp can have a large
	//             multiple of m in it (although I'm not 100% sure
	//             why it can't after this redc!)
	{
		uint T[N_Size * 2];
		for (int i = 0; i < N_Size; ++i)
		{
			T[i] = R[i];
			T[N_Size + i] = 0;
		}

		// MPN_REDC_1(rp, tp, mp, mn, mi);
#pragma unroll 1
		for (int j = 0; j < N_Size; ++j)
		{
			uint cy = 0;
			uint v = T[j] * mi;
			for (int i = 0; i < N_Size; ++i)
			{
				ulong p = ulong(M[i]) * ulong(v) + cy;
				p += T[i + j];
				T[i + j] = uint(p);
				cy = uint(p >> 32);
			}
			R[j] = cy;
		}

		{
			uint cy = 0;
			for (int i = 0; i < N_Size; ++i)
			{
				ulong a = ulong(R[i]) + cy;
				a += T[i + N_Size];
				R[i] = uint(a);
				cy = uint(a >> 32);
			}

			if (cy != 0)
			{
				int32_t borrow = 0;
				uint last_shifted = 0;
				for (int i = 0; i < N_Size; ++i)
				{
					int64_t a = R[i];
					uint b = (M[i] << shift) | last_shifted;
					last_shifted = M[i] >> (32 - shift);
					a = a - int64_t(b) + borrow;
					R[i] = uint(a);
					borrow = int32_t(a >> 32);
				}
			}
		}
	}

	bool result = true;
	if (R[N_Size - 1] != 0)
	{
		// Compare to m+1
		uint cy = 1;
		for (int i = 0; i < N_Size && result; ++i)
		{
			uint a = M[i] + cy;
			cy = a < M[i];
			if (R[i] != a) result = false;
		}
	}
	else
	{
		// Compare to 1
		result = R[0] == 1;
		for (int i = 1; i < N_Size && result; ++i)
		{
			if (R[i] != 0) result = false;
		}
	}

	is_prime[blockDim.x*blockIdx.x + threadIdx.x] = result;
}

#define DEBUG 0

#define MAX_SOURCE_SIZE (0x100000)

const unsigned char  binvert_limb_table[128] = {
	0x01, 0xAB, 0xCD, 0xB7, 0x39, 0xA3, 0xC5, 0xEF,
	0xF1, 0x1B, 0x3D, 0xA7, 0x29, 0x13, 0x35, 0xDF,
	0xE1, 0x8B, 0xAD, 0x97, 0x19, 0x83, 0xA5, 0xCF,
	0xD1, 0xFB, 0x1D, 0x87, 0x09, 0xF3, 0x15, 0xBF,
	0xC1, 0x6B, 0x8D, 0x77, 0xF9, 0x63, 0x85, 0xAF,
	0xB1, 0xDB, 0xFD, 0x67, 0xE9, 0xD3, 0xF5, 0x9F,
	0xA1, 0x4B, 0x6D, 0x57, 0xD9, 0x43, 0x65, 0x8F,
	0x91, 0xBB, 0xDD, 0x47, 0xC9, 0xB3, 0xD5, 0x7F,
	0x81, 0x2B, 0x4D, 0x37, 0xB9, 0x23, 0x45, 0x6F,
	0x71, 0x9B, 0xBD, 0x27, 0xA9, 0x93, 0xB5, 0x5F,
	0x61, 0x0B, 0x2D, 0x17, 0x99, 0x03, 0x25, 0x4F,
	0x51, 0x7B, 0x9D, 0x07, 0x89, 0x73, 0x95, 0x3F,
	0x41, 0xEB, 0x0D, 0xF7, 0x79, 0xE3, 0x05, 0x2F,
	0x31, 0x5B, 0x7D, 0xE7, 0x69, 0x53, 0x75, 0x1F,
	0x21, 0xCB, 0xED, 0xD7, 0x59, 0xC3, 0xE5, 0x0F,
	0x11, 0x3B, 0x5D, 0xC7, 0x49, 0x33, 0x55, 0xFF
};

#define binvert_limb(inv,n)                                             \
  do {                                                                  \
    mp_limb_t  __n = (n);                                               \
    mp_limb_t  __inv;                                                   \
    assert ((__n & 1) == 1);                                            \
                                                                        \
    __inv = binvert_limb_table[(__n/2) & 0x7F]; /*  8 */                \
    if (GMP_LIMB_BITS > 8)   __inv = 2 * __inv - __inv * __inv * __n;   \
    if (GMP_LIMB_BITS > 16)  __inv = 2 * __inv - __inv * __inv * __n;   \
    if (GMP_LIMB_BITS > 32)  __inv = 2 * __inv - __inv * __inv * __n;   \
                                                                        \
    assert ((__inv * __n) == 1);                        \
    (inv) = __inv;                                      \
  } while (0)

static void setup_fermat(int N_Size, int num, const mp_limb_t* M, mp_limb_t* MI, mp_limb_t* R)
{
	assert(N_Size <= MAX_N_SIZE);
	for (int j = 0; j < num; ++j)
	{
		mp_size_t mn = N_Size;
		mp_limb_t mshifted[MAX_N_SIZE];
		mp_srcptr mp;
		mp_ptr rp;
		struct gmp_div_inverse minv;

		// REDCify: r = B^n * 2 % M
		mp = &M[j*N_Size];
		rp = &R[j*N_Size];
		mpn_div_qr_invert(&minv, mp, mn);

		if (minv.shift > 0)
		{
			mpn_lshift(mshifted, mp, mn, minv.shift);
			mp = mshifted;
		}

		for (size_t i = 0; i < mn + 4; ++i) rp[i] = 0;
		rp[mn + 4] = 1 << minv.shift;
		mpn_div_r_preinv_ns(rp, mn + 5, mp, mn, &minv);

		if (minv.shift > 0)
		{
			mpn_rshift(rp, rp, mn, minv.shift);
			mp = &M[j*N_Size];
		}

		mp_limb_t mi;
		binvert_limb(mi, mp[0]);
		MI[j] = -mi;
	}
}

#if DEBUG
#define DPRINTF(fmt, args...) do { printf("line %d: " fmt, __LINE__, ##args); fflush(stdout); } while(0)
#else
#define DPRINTF(fmt, ...) do { } while(0)
#endif

PrimeTestCxt* primeTestInit()
{
	hipError_t cudaStatus;

	PrimeTestCxt* cxt = new PrimeTestCxt;

	int device;
	hipError_t cuResult;
	cuResult = hipInit(0);
	if (cuResult != hipSuccess) {
		printf("hipInit failed!");
		abort();
	}
	cuResult = hipDeviceGet(&device, 0);
	if (cuResult != hipSuccess) {
		printf("hipDeviceGet failed!");
		abort();
	}
	cuResult = hipCtxCreate(&cxt->cudaCxt, hipDeviceScheduleBlockingSync, device);
	if (cuResult != hipSuccess) {
		printf("hipCtxCreate failed!");
		abort();
	}

	// Create memory buffers on the device for each vector 
	cudaStatus = hipMalloc((void**)&cxt->m_mem_obj, MAX_JOB_SIZE * MAX_N_SIZE * sizeof(uint));
	cudaStatus = hipMalloc((void**)&cxt->mi_mem_obj, max(MAX_JOB_SIZE_PRIME, MAX_JOB_SIZE_MOD) * sizeof(uint));
	cudaStatus = hipMalloc((void**)&cxt->r_mem_obj, max(MAX_JOB_SIZE_PRIME * MAX_N_SIZE + 5, MAX_JOB_SIZE_MOD * 6) * sizeof(uint));
	cudaStatus = hipMalloc((void**)&cxt->is_prime_mem_obj, max(MAX_JOB_SIZE_PRIME, MAX_JOB_SIZE_MOD * 2) * sizeof(uint));

	cudaStatus = hipEventCreateWithFlags(&cxt->cudaEvent, hipEventBlockingSync);

	// Create buffers on host
	hipHostMalloc((void**)&cxt->R, sizeof(uint)*max(MAX_N_SIZE*MAX_JOB_SIZE + 5, MAX_JOB_SIZE_MOD * 6));
	hipHostMalloc((void**)&cxt->MI, sizeof(uint)*max(MAX_JOB_SIZE_PRIME, MAX_JOB_SIZE_MOD));
	hipHostMalloc((void**)&cxt->is_prime, sizeof(uint)*max(MAX_JOB_SIZE_PRIME, MAX_JOB_SIZE_MOD * 2));

	hipCtxPopCurrent(NULL);

	return cxt;
}

void primeTest(PrimeTestCxt* cxt, int N_Size, int listSize, const uint* M, uint* is_prime, void (*workFn)(void*), void* workCxt)
{
	std::lock_guard<std::mutex> lock(cxt->cudaMutex);
	hipError_t cudaStatus;
	hipError_t cuResult;

	cuResult = hipCtxPushCurrent(cxt->cudaCxt);
	if (cuResult != hipSuccess) {
		printf("hipCtxPushCurrent failed!");
		abort();
	}

	if (N_Size < 8 || N_Size > MAX_N_SIZE)
	{
		printf("N Size out of bounds\n");
		abort();
	}

	int nextJobSize = min(MAX_JOB_SIZE, listSize);
	int jobSize = 0;
	int lastJobSize = 0;

	if (nextJobSize > 0)
	{
		setup_fermat(N_Size, nextJobSize, M, cxt->MI, cxt->R);
	}

	while (nextJobSize > 0)
	{
		lastJobSize = jobSize;
		jobSize = nextJobSize;
		listSize -= jobSize;
		nextJobSize = min(MAX_JOB_SIZE, listSize);

		// Copy the lists A and B to their respective memory buffers
		cudaStatus = hipMemcpyAsync(cxt->mi_mem_obj, cxt->MI, jobSize * sizeof(uint), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpyAsync(cxt->r_mem_obj, cxt->R, jobSize * N_Size * sizeof(uint), hipMemcpyHostToDevice);
		hipEventRecord(cxt->cudaEvent);
		cudaStatus = hipMemcpyAsync(cxt->m_mem_obj, M, jobSize * N_Size * sizeof(uint), hipMemcpyHostToDevice);

		int blockSize = 1;
		int numBlocks = jobSize;
		while (blockSize < MAX_BLOCK_SIZE && ((numBlocks & 1) == 0))
		{
			numBlocks >>= 1;
			blockSize <<= 1;
		}

		DPRINTF("before execution\n");
#define TEST(N) case N: fermat_test<N> << <numBlocks, blockSize >> >(cxt->m_mem_obj, cxt->mi_mem_obj, cxt->r_mem_obj, cxt->is_prime_mem_obj); break
		switch (N_Size)
		{
			TEST(8);
			TEST(9);
			TEST(10);
			TEST(11);
			TEST(12);
			TEST(13);
			TEST(14);
			TEST(15);
			TEST(16);
			TEST(17);
			TEST(18);
			TEST(19);
			TEST(20);
			TEST(21);
			TEST(22);
			TEST(23);
			TEST(24);
			TEST(25);
			TEST(26);
			TEST(27);
			TEST(28);
			TEST(29);
			TEST(30);
			TEST(31);
			TEST(32);
			TEST(33);
			TEST(34);
			TEST(35);
			TEST(36);
			TEST(37);
			TEST(38);
			TEST(39);
			TEST(40);
			TEST(41);
			TEST(42);
			TEST(43);
			TEST(44);
			TEST(45);
			TEST(46);
			TEST(47);
			TEST(48);
#if 0
			TEST(49);
			TEST(50);
			TEST(51);
			TEST(52);
			TEST(53);
			TEST(54);
			TEST(55);
			TEST(56);
			TEST(57);
			TEST(58);
			TEST(59);
			TEST(60);
			TEST(61);
			TEST(62);
			TEST(63);
			TEST(64);
			TEST(65);
			TEST(66);
			TEST(67);
			TEST(68);
			TEST(69);
			TEST(70);
			TEST(71);
			TEST(72);
			TEST(73);
			TEST(74);
			TEST(75);
			TEST(76);
			TEST(77);
			TEST(78);
			TEST(79);
			TEST(80);
			TEST(81);
			TEST(82);
			TEST(83);
			TEST(84);
			TEST(85);
			TEST(86);
			TEST(87);
			TEST(88);
			TEST(89);
			TEST(90);
			TEST(91);
			TEST(92);
			TEST(93);
			TEST(94);
			TEST(95);
			TEST(96);
			TEST(97);
			TEST(98);
			TEST(99);
			TEST(100);
			TEST(101);
			TEST(102);
			TEST(103);
			TEST(104);
			TEST(105);
			TEST(106);
			TEST(107);
			TEST(108);
			TEST(109);
			TEST(110);
			TEST(111);
			TEST(112);
			TEST(113);
			TEST(114);
			TEST(115);
			TEST(116);
			TEST(117);
			TEST(118);
			TEST(119);
			TEST(120);
			TEST(121);
			TEST(122);
			TEST(123);
			TEST(124);
			TEST(125);
			TEST(126);
			TEST(127);
#endif
		default: abort();
		}
		
#if 1
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			printf("Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			break;
		}
#endif
		if (workFn)
		{
			workFn(workCxt);
			workFn = NULL;
		}

		hipEventSynchronize(cxt->cudaEvent);
		if (cudaStatus != hipSuccess) {
			printf("Sync failed: %s\n", hipGetErrorString(cudaStatus));
			break;
		}

		if (lastJobSize > 0)
		{
			memcpy(is_prime, cxt->is_prime, lastJobSize * sizeof(uint));
			is_prime += lastJobSize;
		}

		if (nextJobSize > 0)
		{
			M += jobSize*N_Size;
			setup_fermat(N_Size, nextJobSize, M, cxt->MI, cxt->R);

			cudaStatus = hipMemcpyAsync(cxt->is_prime, cxt->is_prime_mem_obj, jobSize * sizeof(uint), hipMemcpyDeviceToHost);
		}
		else
		{
			cuResult = hipCtxSynchronize();
			if (cuResult != hipSuccess) {
				printf("hipCtxSynchronize failed!");
			}

			cudaStatus = hipMemcpy(is_prime, cxt->is_prime_mem_obj, jobSize * sizeof(uint), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				printf("Final memcpy failed: %s\n", hipGetErrorString(cudaStatus));
			}
		}
	}

	cuResult = hipCtxPopCurrent(NULL);
	if (cuResult != hipSuccess) {
		printf("hipCtxPopCurrent failed!");
		abort();
	}
}

void primeTestTerm(PrimeTestCxt* cxt)
{
	hipCtxPushCurrent(cxt->cudaCxt);
	hipFree(cxt->mi_mem_obj);
	hipFree(cxt->m_mem_obj);
	hipFree(cxt->r_mem_obj);
	hipFree(cxt->is_prime_mem_obj);

	hipHostFree(cxt->R);
	hipHostFree(cxt->MI);
	hipHostFree(cxt->is_prime);
	delete cxt;

	hipCtxDestroy(cxt->cudaCxt);

	hipDeviceReset();
}
